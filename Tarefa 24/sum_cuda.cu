/*

1)

real    0m0,235s
user    0m0,059s
sys     0m0,155s

real    0m0,253s
user    0m0,067s
sys     0m0,167s

real    0m0,270s
user    0m0,075s
sys     0m0,166s

real    0m0,266s
user    0m0,091s
sys     0m0,152s

real    0m0,239s
user    0m0,073s
sys     0m0,143s

2)

==123303== NVPROF is profiling process 123303, command: ./a.out

Sum = 799999980000000.000000
==123303== Profiling application: ./a.out
==123303== Profiling result:
            Type  Time(%)      Time     Calls       Avg       Min       Max  Name
 GPU activities:   82.35%  41.041ms         1  41.041ms  41.041ms  41.041ms  [CUDA memcpy HtoD]
                   17.60%  8.7735ms         1  8.7735ms  8.7735ms  8.7735ms  sum_cuda(double*, double*, int)

3)

real    0m0,068s
user    0m0,067s
sys     0m0,001s

real    0m0,062s
user    0m0,058s
sys     0m0,004s

real    0m0,043s
user    0m0,042s
sys     0m0,000s

real    0m0,082s
user    0m0,078s
sys     0m0,001s

real    0m0,063s
user    0m0,060s
sys     0m0,000s

4)

real    0m0,041s
user    0m0,187s
sys     0m0,004s

real    0m0,029s
user    0m0,152s
sys     0m0,000s

real    0m0,044s
user    0m0,139s
sys     0m0,004s

real    0m0,009s
user    0m0,048s
sys     0m0,004s

real    0m0,057s
user    0m0,296s
sys     0m0,004s

5)

real    0m0,032s
user    0m0,142s
sys     0m0,004s

real    0m0,065s
user    0m0,207s
sys     0m0,004s

real    0m0,042s
user    0m0,205s
sys     0m0,004s

real    0m0,058s
user    0m0,277s
sys     0m0,003s

real    0m0,070s
user    0m0,323s
sys     0m0,003s

6)

real    0m0,288s
user    0m0,101s
sys     0m0,163s

real    0m0,284s
user    0m0,089s
sys     0m0,171s

real    0m0,265s
user    0m0,090s
sys     0m0,154s

real    0m0,280s
user    0m0,074s
sys     0m0,186s

real    0m0,273s
user    0m0,106s
sys     0m0,146s

==125067== NVPROF is profiling process 125067, command: ./a.out

Sum = 799999980000000.000000
==125067== Profiling application: ./a.out
==125067== Profiling result:
            Type  Time(%)      Time     Calls       Avg       Min       Max  Name
 GPU activities:   83.24%  49.618ms         1  49.618ms  49.618ms  49.618ms  [CUDA memcpy HtoD]
                   16.71%  9.9626ms         1  9.9626ms  9.9626ms  9.9626ms  sum_cuda(double*, double*, int)

Coment??rios:

Henrique, n??s tentamos usar o servidor da PUC para pegar as m??tricas do CUDA com o comando nvprof, mas ele n??o listou tudo
que a gente precisava. Acreditamos que tem a ver com a vers??o do driver, porque testamos no PC de um dos integrantes do grupo
que tem uma GPU Nvidia com a vers??o mais nova do driver e funcionou. A GPU dele ?? uma Nvidia GTX 1060 6GB e a vers??o 
do driver ?? 470.74-10. Por causa disso, acreditamos que seria mais justo testar a vers??o OpenMP para GPU na mesma placa tamb??m.

A vers??o em CUDA rodou bem pior que as outras vers??es, e o grupo acredita que isso se deve ?? complexidade do c??digo comparada 
?? outras vers??es. Enquanto o c??digo em CUDA precisa chamar uma fun????o, rodar diversos loops for e fazer v??rias opera????es, as
outras vers??es s??o bem mais simples, tendo apenas 1 loop for.

Dentre as vers??es com e sem __shared__, podemos notar uma melhora consider??vel. A vers??o com uso do __shared__ tem uma melhora
enorme, pois o uso dessa flag faz uso da mem??ria cache interna dos blocos da GPU, que ?? bem mais r??pida que a mem??ria global.

Entre as duas vers??es do c??digo em CUDA, o maior overhead ?? causado entre a transfer??ncia dos dados entre a mem??ria global da GPU
para as threads individuais. Como o uso da flag __shared__ favorece o uso da cache dos blocos de thread da GPU, esse overhead ??
diminu??do.

Obs: deixamos todos os c??digos utilizados neste arquivo.

*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void sum_cuda(double* a, double *s, int width) {
  int t = threadIdx.x;
  int b = blockIdx.x*blockDim.x;

  __shared__ double o[1024];  

  if(b+t < width)
    o[t] = a[b+t];

  __syncthreads();
  
  int i;
  for(i = blockDim.x/2; i > 0; i /= 2) {
    if(t < i && b+t+i < width)
      o[t] += o[t+i];
    
    __syncthreads();
  }

  if(t == 0)
    s[blockIdx.x] = o[0];  
} 

int main()
{
  int width = 40000000;
  int size = width * sizeof(double);

  int block_size = 1024;
  int num_blocks = (width-1)/block_size+1;
  int s_size = (num_blocks * sizeof(double));  
 
  double *a = (double*) malloc (size);
  double *s = (double*) malloc (s_size);

  for(int i = 0; i < width; i++)
    a[i] = i;

  double *d_a, *d_s;

  // aloca????o e c??pia dos dados
  hipMalloc((void **) &d_a, size);
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

  hipMalloc((void **) &d_s, s_size);

  // defini????o do n??mero de blocos e threads
  dim3 dimGrid(num_blocks,1,1);
  dim3 dimBlock(block_size,1,1);

  // chamada do kernel
  sum_cuda<<<dimGrid,dimBlock>>>(d_a, d_s, width);

  // c??pia dos resultados para o host
  hipMemcpy(s, d_s, s_size, hipMemcpyDeviceToHost);

  // soma das redu????es parciais
  for(int i = 1; i < num_blocks; i++) 
    s[0] += s[i];

  printf("\nSum = %f\n",s[0]);
  
  hipFree(d_a);
  hipFree(d_s);
}

/* sum-cuda-noshared
#include <stdio.h>
#include <stdlib.h>

__global__ void sum_cuda(double* a, double *s, int width) {
  int t = threadIdx.x;
  int b = blockIdx.x*blockDim.x;
 
  int i;
  for(i = blockDim.x/2; i > 0; i /= 2) {
    if(t < i && b+t+i < width)
      a[t+b] += a[t+b+i];
    
    __syncthreads();
  }
  if(t == 0)
    s[blockIdx.x] = a[t+b];  
} 

int main()
{
  int width = 40000000;
  int size = width * sizeof(double);

  int block_size = 1024;
  int num_blocks = (width-1)/block_size+1;
  int s_size = (num_blocks * sizeof(double));  
 
  double *a = (double*) malloc (size);
  double *s = (double*) malloc (s_size);

  for(int i = 0; i < width; i++)
    a[i] = i;

  double *d_a, *d_s;

  // aloca????o e c??pia dos dados
  cudaMalloc((void **) &d_a, size);
  cudaMemcpy(d_a, a, size, cudaMemcpyHostToDevice);

  cudaMalloc((void **) &d_s, s_size);

  // defini????o do n??mero de blocos e threads
  dim3 dimGrid(num_blocks,1,1);
  dim3 dimBlock(block_size,1,1);

  // chamada do kernel
  sum_cuda<<<dimGrid,dimBlock>>>(d_a, d_s, width);

  // c??pia dos resultados para o host
  cudaMemcpy(s, d_s, s_size, cudaMemcpyDeviceToHost);

  // soma das redu????es parciais
  for(int i = 1; i < num_blocks; i++) 
    s[0] += s[i];

  printf("\nSum = %f\n",s[0]);
  
  cudaFree(d_a);
  cudaFree(d_s);
}
*/

/* sum sequencial
#include <stdio.h>
#include <stdlib.h>

int main()
{
  int width = 40000000;
  double sum = 0;


  for(int i = 0; i < width; i++)
  {
     sum += i;
  }

  printf("\nSum = %f\n",sum);
}
*/

/* sum paralelo
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>


int main()
{
  double sum   =  0;
  int width    =  40000000;

  #pragma omp parallel for simd reduction(+:sum)
  for(int i = 0; i < width; i++)
  {
     sum += i;
  }

  printf("\nSum = %lf\n",sum);
}
*/

/* sum paralelo gpu
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>


int main()
{
  double sum   =  0;
  int width    =  40000000;

  #pragma omp target teams distribute parallel for simd map(tofrom:sum) map(to:width) reduction(+:sum) 
  for(int i = 0; i < width; i++)
  {
     sum += i;
  }

  printf("\nSum = %lf\n",sum);
}
*/
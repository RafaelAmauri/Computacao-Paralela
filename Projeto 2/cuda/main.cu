﻿#include "hip/hip_runtime.h"
#include ""
#include "headers/Blockchain.cuh"

#include <stdio.h>

using namespace std;

int main()
{
	uint32_t nDifficulty = 5;
	Blockchain bChain = Blockchain(nDifficulty);

	cout << "Mining block 1..." << endl;
	bChain.AddBlock(new Block(1, "Block 1 Data"));

	cout << "Mining block 2..." << endl;
	bChain.AddBlock(new Block(2, "Block 2 Data"));

	cout << "Mining block 3..." << endl;
	bChain.AddBlock(new Block(3, "Block 3 Data"));

	//bChain.PrintBlockchain();
	return 0;
}
#include "hip/hip_runtime.h"
#include<algorithm>
#include "../headers/Block.cuh"
#include "../headers/Sha256.cuh"
#include "../headers/Sha256Host.cuh"

#define THREADS 1500
#define BLOCKS 256
#define DIFFICULTY 3
#define RANDOM_LEN 20

typedef unsigned char BYTE;

__constant__ BYTE characterSet[63] = { "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ1234567890" };

__device__ unsigned long long deviceRandomGen(unsigned long long x)
{
    x ^= (x << 21);
    x ^= (x >> 35);
    x ^= (x << 4);
    return x;
}

__global__ void sha256_cuda(BYTE* input, BYTE* solution, int* blockContainsSolution, unsigned long long baseSeed, size_t textLen)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long seed = baseSeed;
    seed += (unsigned long long)i;

    BYTE digest[32];
    BYTE random[RANDOM_LEN];
    memset(digest, 0, SHA256::DIGEST_SIZE);
    memset(random, 0, RANDOM_LEN);

    for (int j = 0; j < RANDOM_LEN; j++)
    {
        seed = deviceRandomGen(seed);
        int randomIdx = (int)(seed % 62);
        random[j] = characterSet[randomIdx];
    }

    SHA256CUDA::sha256(input, random, textLen, RANDOM_LEN, digest);

    for (int j = 0; j < DIFFICULTY; j++)
    {
        if (digest[j] > 0)
        {
            return;
        }
    }

    if (*blockContainsSolution == 1)
    {
        return;
    }

    *blockContainsSolution = 1;

    for (int j = 0; j < RANDOM_LEN; j++)
    {
        solution[j] = random[j];
    }
}

__global__ void initSolutionMemory(int* blockContainsSolution)
{
    *blockContainsSolution = -1;
}

Block::Block(uint32_t nIndexIn, const string& sDataIn)
{
    _sNonce = "-1";
    _nIndex = nIndexIn;
    _tTime = time(nullptr);
    _sData = sDataIn;
    _sHash = _CalculateHash();
}

string& Block::GetHash()
{
    return _sHash;
}

void Block::hostRandomGen(unsigned long long* x)
{
    *x ^= (*x << 21);
    *x ^= (*x >> 35);
    *x ^= (*x << 4);
}

int Block::MineBlock(uint32_t nDifficulty)
{
    bool solution = false;

    stringstream ss;
    ss << _nIndex << _tTime << _sData << sPrevHash;

    string x = ss.str();

    BYTE* d_input;
    BYTE* input = (BYTE*)x.c_str();
    hipMalloc(&d_input, sizeof(BYTE) * x.length());
    hipMemcpy(d_input, input, sizeof(BYTE) * x.length(), hipMemcpyHostToDevice);

    BYTE* blockSolution = (BYTE*)malloc(sizeof(BYTE) * RANDOM_LEN);
    BYTE* d_solution;
    hipMalloc(&d_solution, sizeof(BYTE) * RANDOM_LEN);

    int* blockContainsSolution = (int*)malloc(sizeof(int));
    int* d_blockContainsSolution;
    hipMalloc(&d_blockContainsSolution, sizeof(int));

    unsigned long long rngSeed = static_cast<unsigned long long>(time(nullptr));

    initSolutionMemory<<<1, 1>>>(d_blockContainsSolution);

    while (!solution) {
        hostRandomGen(&rngSeed);

        sha256_cuda<<<THREADS, BLOCKS>>>(d_input, d_solution, d_blockContainsSolution, rngSeed, x.length());
        hipDeviceSynchronize();

        hipMemcpy(blockContainsSolution, d_blockContainsSolution, sizeof(int), hipMemcpyDeviceToHost);

        if (*blockContainsSolution == 1)
        {
            hipMemcpy(blockSolution, d_solution, sizeof(BYTE) * RANDOM_LEN, hipMemcpyDeviceToHost);
            solution = true;
            
            break;
        }
    }

    hipDeviceReset();
    _sNonce = string((const char*)blockSolution);
    _sHash = _CalculateHash();

    return NULL;
}
    
void Block::PrintBlock()
{
    cout << "-----------" << "Block " << _nIndex << "-----------" << "\n";
    cout << "Nonce: " << _sNonce << "\n";
    cout << "Hash: " << _sHash << "\n";
    cout << "Previous Hash: " << sPrevHash << "\n";
    cout << "Data: " << _sData << "\n";
    cout << "Time: " << _tTime << "\n";
}

inline string Block::_CalculateHash() const
{
    stringstream ss;
    ss << _nIndex << _tTime << _sData << sPrevHash << _sNonce;

    return sha256(ss.str());
}
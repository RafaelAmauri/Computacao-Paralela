
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void scan_cuda(double* a, double *s, int width) {
 int t = threadIdx.x;
 int b = blockIdx.x*blockDim.x; 
 double x;

 __shared__ double p[1024];

 if(b+t < width)
     p[t] = a[b+t];

 __syncthreads();

 for(int i = 1; i < blockDim.x; i *= 2) {
      if(t >= i)
          x = p[t] + p[t-i];

      __syncthreads();

      if(t >= i)
          p[t] = x;

      __syncthreads();
  }

 if(b + t < width)
     a[b+t] = p[t];

 if(t == blockDim.x-1)
     s[blockIdx.x+1] = a[b+t];

} 

__global__ void add_cuda(double *a, double *s, int width) {
  int t = threadIdx.x;                                                                                                           
  int b = blockIdx.x*blockDim.x; 
 
  if(b+t < width) 
      a[b+t] += s[blockIdx.x];
}




int main()
{
  int width = 40000000;
  int size = width * sizeof(double);

  int block_size = 1024;
  int num_blocks = (width-1)/block_size+1;
  int s_size = (num_blocks * sizeof(double));  
 
  double *a = (double*) malloc (size);
  double *s = (double*) malloc (s_size);

  for(int i = 0; i < width; i++)
    a[i] = i;

  double *d_a, *d_s;

  s[0] = 0;
  for (int i = 1; i < num_blocks; i++)
    s[i] += s[i-1];

  printf("\na[%d] = %f\n",width-1,a[width-1]);
  
  hipFree(d_a);
  hipFree(d_s);
}
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "constants.h"
__global__ void k_means_kernel_assignment_split(const float *imageIn, float *cluster, float *centroids, const int means)
{
    // do this for each individual pixel
    float min_temp = BIGNUM;
    float distance;
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int j;
    for (j = 0; j <means;j++) {
                distance = fabs(centroids[j]-imageIn[index]);// compare image to centroids
                if (distance<min_temp){
                    min_temp = distance;
		    cluster[index]= j;		    
		}
   }
}



__global__ void k_means_kernel_writeBack_split(float *imageOut, const float *imageIn, const float *centroids, const float *cluster)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	int temp2 = (int)cluster[index];
	imageOut[index] = centroids[temp2];
}

float* k_means_split(float *imageIn, int clusters, int dimension, int iterations){
   struct timespec diff(struct timespec start, struct timespec end);
   struct timespec timeStart, timeEnd;
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    
    // the cluster vector
    int numElements = (dimension)*(dimension);
    size_t size = numElements * sizeof(float);
    float *cluster = (float*) malloc(size);// which centroid does each cluster belong to?
    float *imageOut = (float*) malloc(size);//output image

    // the centroids or means
    int means = clusters;
    size_t size2= means * sizeof(float);
    float *centroids = (float*) malloc(size2);// list of centroids(means)
	float *accumulator = (float*) malloc(size2);
	float *numPixelsCentroid = (float*) malloc(size2);/*needed for the update average step*/
     
    float range = 255/(means-1);
    //initialize step to set everything to zero
    for (int m = 0; m < means; m++) {
        centroids[m] = range*m;
        accumulator[m] =0;
        numPixelsCentroid[m] =0;
    }

    // Allocat DEVICE vectors
    float *d_imageIn = NULL;
    err = hipMalloc((void**)&d_imageIn,size);
    if(err != hipSuccess)
    {
	fprintf(stderr, "Failed to allocate device vector imageIn (error code %s)!\n", hipGetErrorString(err));
	exit(EXIT_FAILURE);
    }
    float *d_imageOut = NULL;
    err = hipMalloc((void**)&d_imageOut,size);
    if(err != hipSuccess)
    {
	fprintf(stderr, "Failed to allocate device vector imageOut (error code %s)!\n", hipGetErrorString(err));
	exit(EXIT_FAILURE);
    }
    float *d_cluster = NULL;
    err = hipMalloc((void**)&d_cluster,size);
    if(err != hipSuccess)
    {
	fprintf(stderr, "Failed to allocate device vector cluster (error code %s)!\n", hipGetErrorString(err));
	exit(EXIT_FAILURE);
    }
    float *d_centroids = NULL;
    err = hipMalloc((void**)&d_centroids,size2);
    if(err != hipSuccess)
    {
	fprintf(stderr, "Failed to allocate device vector centroids (error code %s)!\n", hipGetErrorString(err));
	exit(EXIT_FAILURE);
    }


    // Copy vectors to DEVICE
    printf("Copy input data from the host memory to the CUDA device \n");
      
    err = hipMemcpy(d_imageIn, imageIn, size , hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
	fprintf(stderr, "Failed to copy Vector imagein from host to device (error code %s)! \n", hipGetErrorString(err));
    } 
    err = hipMemcpy(d_imageOut, imageOut, size , hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
	fprintf(stderr, "Failed to copy Vector imageOut from host to device (error code %s)! \n", hipGetErrorString(err));
    } 
    err = hipMemcpy(d_cluster, cluster, size , hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
	fprintf(stderr, "Failed to copy Vector cluster from host to device (error code %s)! \n", hipGetErrorString(err));
    }   

    err = hipMemcpy(d_centroids, centroids, size2 , hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
	fprintf(stderr, "Failed to copy Vector centroids from host to device (error code %s)! \n", hipGetErrorString(err));
    }

    // Launch the kmeans CUDA kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + 256 - 1)/256;

    dim3 dimBlock(THREADS_PER_BLOCK);
    dim3 dimGrid(BLOCKS_PER_GRID,CLUSTERS,1);
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &timeStart);

for (int iters =0; iters<iterations; iters++){
    	k_means_kernel_assignment_split<<<dimGrid,dimBlock>>>(d_imageIn, d_cluster, d_centroids, means);
        hipDeviceSynchronize();

    err = hipGetLastError();

    if( err != hipSuccess)
    {
	fprintf(stderr, "Failed to launch kmeans kernel (error code %s)! \n",hipGetErrorString(err));
	exit(EXIT_FAILURE);
    }
	err = hipMemcpy(centroids,d_centroids,size2,hipMemcpyDeviceToHost);
	err = hipMemcpy(cluster, d_cluster, size , hipMemcpyDeviceToHost);
    if(err != hipSuccess)
    {
	fprintf(stderr, "Failed to copy from host to device (error code %s)! \n", hipGetErrorString(err));
    }   
  	 for ( int h = 0; h < means; h++) {
	    for ( int i=0; i < numElements-1; i++){
	    if (cluster[i] == h){
	    int temp1 = (int)cluster[i];
	    accumulator[temp1] += imageIn[i];
	    numPixelsCentroid[temp1]+=1;
	    }
	   }
 	if (numPixelsCentroid[h] != 0){
            centroids[h] = accumulator[h]/numPixelsCentroid[h];
            //reset
	    }
            accumulator[h] = 0;
            numPixelsCentroid[h] =0;

        }
    err = hipMemcpy(d_centroids, centroids, size2 , hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
	fprintf(stderr, "Failed to copy Vector centroids from host to device (error code %s)! \n", hipGetErrorString(err));
    }
	iters++;

}
 k_means_kernel_writeBack_split<<<blocksPerGrid,threadsPerBlock>>>(d_imageOut, d_imageIn, d_centroids, d_cluster);
 hipDeviceSynchronize();  
 clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &timeEnd);
printf("Internal GPU time: %ld \n", (long int)(((double)GIG *diff(timeStart,timeEnd).tv_sec + diff(timeStart,timeEnd).tv_nsec)));
  err = hipMemcpy(centroids,d_centroids,size2,hipMemcpyDeviceToHost);
    if(err != hipSuccess)
    {
	fprintf(stderr, "Failed to copy vector centroids from device to host (error code %s)! \n",hipGetErrorString(err));
	exit(EXIT_FAILURE);
    }
    err = hipMemcpy(imageOut,d_imageOut,size,hipMemcpyDeviceToHost);
    if(err != hipSuccess)
    {
	fprintf(stderr, "Failed to copy vector imageOut from device to host (error code %s)! \n",hipGetErrorString(err));
	exit(EXIT_FAILURE);
    }
    // Free divice global memory
    err = hipFree(d_imageOut);
    if(err != hipSuccess)
    {
	fprintf(stderr,"Failed to free device vector centroids (error code %s)! \n",hipGetErrorString(err));
	exit(EXIT_FAILURE);
    }
    err = hipFree(d_centroids);
    if(err != hipSuccess)
    {
	fprintf(stderr,"Failed to free device vector centroids (error code %s)! \n",hipGetErrorString(err));
	exit(EXIT_FAILURE);
    }

    
    err = hipFree(d_imageIn);
    if(err != hipSuccess)
    {
	fprintf(stderr,"Failed to free device vector imageIn (error code %s)! \n",hipGetErrorString(err));
	exit(EXIT_FAILURE);
    }

    err = hipFree(d_cluster);
    if(err != hipSuccess)
    {
	fprintf(stderr,"Failed to free device vector cluster (error code %s)! \n",hipGetErrorString(err));
	exit(EXIT_FAILURE);
    }

    // Reset the device and exit
    err = hipDeviceReset();
    if(err != hipSuccess)
    {
	fprintf(stderr,"Failed to deinitialize the device! (error code %s)! \n",hipGetErrorString(err));
	exit(EXIT_FAILURE);
    }
   // set output
   for (int m = 0; m < means; m++) {
	printf("%f \n",centroids[m]);
}
   return imageOut;
}

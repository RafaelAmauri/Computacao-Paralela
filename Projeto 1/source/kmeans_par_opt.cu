#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "constants.h"
__global__ void k_means_kernel_assignment_opt(const float *imageIn, float *cluster, float *centroids, const int means,float *accumulator,float *numPixelsCentroid)
{
    __shared__ float partialAccumulator[CLUSTERS][THREADS_PER_BLOCK];
    __shared__ float partialNumPixelsCentroid[CLUSTERS][THREADS_PER_BLOCK];
    // do this for each individual pixel
    float min_temp = BIGNUM;
    float distance;
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int threadNum = threadIdx.x;
    int j,k;
    for (j = 0; j <means;j++) {
                distance = fabs(centroids[j]-imageIn[index]);// compare image to centroids
                if (distance<min_temp){
                    min_temp = distance;
		    cluster[index]= j;		    
		}
	}

  int temp1 = (int)cluster[index];
  partialAccumulator[temp1][threadNum] = imageIn[index];
  partialNumPixelsCentroid[temp1][threadNum] = 1;
  __syncthreads();
  if ( threadNum< CLUSTERS){
  	for (k=1; k<THREADS_PER_BLOCK;k++){
		partialAccumulator[threadNum][0] += partialAccumulator[threadNum][k];
		partialNumPixelsCentroid[threadNum][0] += partialNumPixelsCentroid[threadNum][k];
	}
  __syncthreads();
	accumulator[threadNum*(int)BLOCKS_PER_GRID+blockIdx.x] += partialAccumulator[threadNum][0];
	numPixelsCentroid[threadNum*(int)BLOCKS_PER_GRID+blockIdx.x] += partialNumPixelsCentroid[threadNum][0];
  }

  __syncthreads();
  
}


__global__ void k_means_kernel_update_opt(const float *imageIn, float *cluster,float *centroids,float *accumulator,float *numPixelsCentroid, int numElements)
{
	int index = threadIdx.x;
	int i;
	for (i =1; i<(int)BLOCKS_PER_GRID; i++){
		accumulator[index*(int)BLOCKS_PER_GRID] += accumulator[index*(int)BLOCKS_PER_GRID+i];
		numPixelsCentroid[index*(int)BLOCKS_PER_GRID] += numPixelsCentroid[index*(int)BLOCKS_PER_GRID+i];
	
	accumulator[index*(int)BLOCKS_PER_GRID+i] = 0;
	numPixelsCentroid[index*(int)BLOCKS_PER_GRID+i] = 0;
	}

	if (numPixelsCentroid[index*(int)BLOCKS_PER_GRID] != 0){
            centroids[index] =  accumulator[index*(int)BLOCKS_PER_GRID]/numPixelsCentroid[index*(int)BLOCKS_PER_GRID];
        }
	    numPixelsCentroid[index*(int)BLOCKS_PER_GRID]= 0;
	    accumulator[index*(int)BLOCKS_PER_GRID] = 0;
}

__global__ void k_means_kernel_writeBack_opt(float *imageOut, const float *imageIn, const float *centroids, const float *cluster)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	int temp2 = (int)cluster[index];
	imageOut[index] = centroids[temp2];
}

float* k_means_parallel_optimized(float *imageIn, int clusters, int dimension, int iterations){
   struct timespec diff(struct timespec start, struct timespec end);
   struct timespec timeStart, timeEnd;
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    
    // the cluster vector
    int numElements = (dimension)*(dimension);
    size_t size = numElements * sizeof(float);
    float *cluster = (float*) malloc(size);// which centroid does each cluster belong to?
    float *imageOut = (float*) malloc(size);//output image

    // the centroids or means
    int means = clusters;
    size_t size2= means * sizeof(float);
    float *centroids = (float*) malloc(size2);// list of centroids(means)
    size_t size3= means * sizeof(float)*(int)((numElements + (256) - 1)/(256));
    float *accumulator = (float*) malloc(size3);
    float *numPixelsCentroid = (float*) malloc(size3);/*needed for the update average step*/
     
    float range = 255/(means-1);
    //initialize step to set everything to zero
    for (int m = 0; m < means; m++) {
        centroids[m] = range*m;
        accumulator[m] =0;
        numPixelsCentroid[m] =0;
    }

   

    // Allocat DEVICE vectors
    float *d_imageIn = NULL;
    err = hipMalloc((void**)&d_imageIn,size);
    if(err != hipSuccess)
    {
	fprintf(stderr, "Failed to allocate device vector imageIn (error code %s)!\n", hipGetErrorString(err));
	exit(EXIT_FAILURE);
    }
    float *d_imageOut = NULL;
    err = hipMalloc((void**)&d_imageOut,size);
    if(err != hipSuccess)
    {
	fprintf(stderr, "Failed to allocate device vector imageOut (error code %s)!\n", hipGetErrorString(err));
	exit(EXIT_FAILURE);
    }
    float *d_cluster = NULL;
    err = hipMalloc((void**)&d_cluster,size);
    if(err != hipSuccess)
    {
	fprintf(stderr, "Failed to allocate device vector cluster (error code %s)!\n", hipGetErrorString(err));
	exit(EXIT_FAILURE);
    }
    float *d_centroids = NULL;
    err = hipMalloc((void**)&d_centroids,size2);
    if(err != hipSuccess)
    {
	fprintf(stderr, "Failed to allocate device vector centroids (error code %s)!\n", hipGetErrorString(err));
	exit(EXIT_FAILURE);
    }
    float *d_accumulator = NULL;
    err = hipMalloc((void**)&d_accumulator,size3);
    if(err != hipSuccess)
    {
	fprintf(stderr, "Failed to allocate device vector accumulator (error code %s)!\n", hipGetErrorString(err));
	exit(EXIT_FAILURE);
    }
    float *d_numPixelsCentroid = NULL;
    err = hipMalloc((void**)&d_numPixelsCentroid,size3);
    if(err != hipSuccess)
    {
	fprintf(stderr, "Failed to allocate device numPixelsCentroid accumulator (error code %s)!\n", hipGetErrorString(err));
	exit(EXIT_FAILURE);
    }


    // Copy vectors to DEVICE
    printf("Copy input data from the host memory to the CUDA device \n");
      
    err = hipMemcpy(d_imageIn, imageIn, size , hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
	fprintf(stderr, "Failed to copy Vector imagein from host to device (error code %s)! \n", hipGetErrorString(err));
    } 
    err = hipMemcpy(d_imageOut, imageOut, size , hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
	fprintf(stderr, "Failed to copy Vector imageOut from host to device (error code %s)! \n", hipGetErrorString(err));
    } 
    err = hipMemcpy(d_cluster, cluster, size , hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
	fprintf(stderr, "Failed to copy Vector cluster from host to device (error code %s)! \n", hipGetErrorString(err));
    }   

    err = hipMemcpy(d_centroids, centroids, size2 , hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
	fprintf(stderr, "Failed to copy Vector centroids from host to device (error code %s)! \n", hipGetErrorString(err));
    }
    err = hipMemcpy(d_accumulator, accumulator, size3 , hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
	fprintf(stderr, "Failed to copy Vector accumulators from host to device (error code %s)! \n",    hipGetErrorString(err));
    }
   err = hipMemcpy(d_numPixelsCentroid, numPixelsCentroid, size3 , hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
	fprintf(stderr, "Failed to copy Vector numPixelsCentroid from host to device (error code %s)! \n",    hipGetErrorString(err));
    }
    // Launch the kmeans CUDA kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + 256 - 1)/256;

    dim3 dimBlock(THREADS_PER_BLOCK);
    dim3 dimGrid(BLOCKS_PER_GRID,CLUSTERS,1);
    dim3 dimBlockVR(int(BLOCKS_PER_GRID/THREADS_PER_BLOCK+.5),CLUSTERS,1);
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &timeStart);

     for (int iters =0; iters<iterations; iters++){

    	k_means_kernel_assignment_opt<<<dimGrid,dimBlock>>>(d_imageIn, d_cluster, d_centroids, means, d_accumulator,d_numPixelsCentroid);
        hipDeviceSynchronize();
         k_means_kernel_update_opt<<<1,means>>>(d_imageIn, d_cluster,d_centroids, d_accumulator,d_numPixelsCentroid,numElements);
	hipDeviceSynchronize();   

}
    k_means_kernel_writeBack_opt<<<blocksPerGrid,threadsPerBlock>>>(d_imageOut, d_imageIn, d_centroids, d_cluster);
    hipDeviceSynchronize();  

    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &timeEnd);

    err = hipGetLastError();

    if( err != hipSuccess)
    {
	fprintf(stderr, "Failed to launch kmeans kernel (error code %s)! \n",hipGetErrorString(err));
	exit(EXIT_FAILURE);
    }
printf("Internal GPU time: %ld \n", (long int)(((double)GIG *diff(timeStart,timeEnd).tv_sec + diff(timeStart,timeEnd).tv_nsec)));
    
    // Copy the device result vector in device memory to the host result vector
    // in host memory
    printf("Copy output data from CUDA device to the host memory \n");

  err = hipMemcpy(centroids,d_centroids,size2,hipMemcpyDeviceToHost);
    if(err != hipSuccess)
    {
	fprintf(stderr, "Failed to copy vector centroids from device to host (error code %s)! \n",hipGetErrorString(err));
	exit(EXIT_FAILURE);
    }
    err = hipMemcpy(imageOut,d_imageOut,size,hipMemcpyDeviceToHost);
    if(err != hipSuccess)
    {
	fprintf(stderr, "Failed to copy vector imageOut from device to host (error code %s)! \n",hipGetErrorString(err));
	exit(EXIT_FAILURE);
    }
    // Free divice global memory
    err = hipFree(d_imageOut);
    if(err != hipSuccess)
    {
	fprintf(stderr,"Failed to free device vector centroids (error code %s)! \n",hipGetErrorString(err));
	exit(EXIT_FAILURE);
    }
    err = hipFree(d_centroids);
    if(err != hipSuccess)
    {
	fprintf(stderr,"Failed to free device vector centroids (error code %s)! \n",hipGetErrorString(err));
	exit(EXIT_FAILURE);
    }
 err = hipFree(d_accumulator);
    if(err != hipSuccess)
    {
	fprintf(stderr,"Failed to free device vector centroids (error code %s)! \n",hipGetErrorString(err));
	exit(EXIT_FAILURE);
    }
 err = hipFree(d_numPixelsCentroid);
    if(err != hipSuccess)
    {
	fprintf(stderr,"Failed to free device vector centroids (error code %s)! \n",hipGetErrorString(err));
	exit(EXIT_FAILURE);
    }
    
    err = hipFree(d_imageIn);
    if(err != hipSuccess)
    {
	fprintf(stderr,"Failed to free device vector imageIn (error code %s)! \n",hipGetErrorString(err));
	exit(EXIT_FAILURE);
    }

    err = hipFree(d_cluster);
    if(err != hipSuccess)
    {
	fprintf(stderr,"Failed to free device vector cluster (error code %s)! \n",hipGetErrorString(err));
	exit(EXIT_FAILURE);
    }

    // Reset the device and exit
    err = hipDeviceReset();
    if(err != hipSuccess)
    {
	fprintf(stderr,"Failed to deinitialize the device! (error code %s)! \n",hipGetErrorString(err));
	exit(EXIT_FAILURE);
    }
   // set output
   for (int m = 0; m < means; m++) {
	printf("%f \n",centroids[m]);
}
   return imageOut;
}
